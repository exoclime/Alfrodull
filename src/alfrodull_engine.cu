#include "alfrodull_engine.h"

alfrodull_engine::alfrodull_engine()
{
  printf("Creating Alfrodull engine\n");
}

void alfrodull_engine::load_opacities(const string & filename)
{
  printf("Loading opacities from %s\n", filename.c_str());

  opacities.load_opacity_table(filename);
}

void alfrodull_engine::init()
{
  printf("Alfrodull Init\n");

  load_opacities("input/opac_sample.h5");
}

void alfrodull_engine::set_parameters(const int & nlayer_,
				      const bool & iso_,
				      const double & T_star_)
{
  nlayer = nlayer_;
  ninterface = nlayer + 1;
  iso = iso_;
  T_star = T_star_;
  
  // TODO: maybe should stay in opacities object
  nbin = opacities.nbin;

  // prepare_planck_table();
}

void alfrodull_engine::allocate_internal_variables()
{
  int nlayer_nbin = nlayer*opacities.nbin;
  int nlayer_plus2_nbin = (nlayer + 2)*opacities.nbin;
  int ninterface_nbin = ninterface*opacities.nbin;
  int nlayer_wg_nbin = nlayer*opacities.ny*opacities.nbin;
  int ninterface_wg_nbin = ninterface*opacities.ny*opacities.nbin;

  // scatter cross section layer and interface
  // those are shared for print out
  scatter_cross_section_lay.allocate(nlayer_nbin);
  scatter_cross_section_inter.allocate(ninterface_nbin);
  planckband_lay.allocate(nlayer_plus2_nbin);
  planckband_int.allocate(ninterface_nbin);

  
  if (iso)
    {
      delta_tau_wg.allocate(nlayer_wg_nbin);
    }
  else
    {
      delta_tau_wg_upper.allocate(nlayer_wg_nbin);
      delta_tau_wg_lower.allocate(nlayer_wg_nbin);
    }
  
  // flux computation internal quantities
  // TODO: not needed to allocate everything, depending on iso or noniso
  if (iso)
    { 
      M_term.allocate(nlayer_wg_nbin);
      N_term.allocate(nlayer_wg_nbin);   
      P_term.allocate(nlayer_wg_nbin);
      G_plus.allocate(nlayer_wg_nbin);
      G_minus.allocate(nlayer_wg_nbin);
      w_0.allocate(nlayer_wg_nbin);
    }
  else
    { 
      M_upper.allocate(nlayer_wg_nbin);
      M_lower.allocate(nlayer_wg_nbin);
      N_upper.allocate(nlayer_wg_nbin);
      N_lower.allocate(nlayer_wg_nbin);
      P_upper.allocate(nlayer_wg_nbin);
      P_lower.allocate(nlayer_wg_nbin);
      G_plus_upper.allocate(nlayer_wg_nbin);
      G_plus_lower.allocate(nlayer_wg_nbin);
      G_minus_upper.allocate(nlayer_wg_nbin);
      G_minus_lower.allocate(nlayer_wg_nbin);
      w_0_upper.allocate(nlayer_wg_nbin);
      w_0_lower.allocate(nlayer_wg_nbin);
    }
}

// return device pointers for helios data save
// TODO: how ugly can it get, really?
void alfrodull_engine::get_device_pointers_for_helios_write(double *& dev_scat_cross_section_lay,
							    double *& dev_scat_cross_section_int,
							    double *& dev_interwave,
							    double *& dev_deltawave,
							    double *& dev_planck_lay,
							    double *& dev_planck_int,
							    double *& dev_planck_grid,
							    double *& dev_delta_tau_wg,
							    double *& dev_delta_tau_wg_upper,
							    double *& dev_delta_tau_wg_lower,
							    int & dim,
							    int & step
							    )
{
  dev_scat_cross_section_lay = *scatter_cross_section_lay;
  dev_scat_cross_section_int = *scatter_cross_section_inter;
  dev_interwave = *opacities.dev_opac_interwave;
  dev_deltawave = *opacities.dev_opac_deltawave;
  dev_planck_lay = *planckband_lay;
  dev_planck_int = *planckband_int;
  dev_planck_grid = *plancktable.planck_grid;
  dev_delta_tau_wg = *delta_tau_wg;
  dev_delta_tau_wg_upper = *delta_tau_wg_upper;
  dev_delta_tau_wg_lower = *delta_tau_wg_lower;
  
  dim = plancktable.dim;
  step = plancktable.step;
}

// TODO: check how to enforce this: must be called after loading opacities and setting parameters
void alfrodull_engine::prepare_planck_table()
{
  plancktable.construct_planck_table(*opacities.dev_opac_interwave,
				     *opacities.dev_opac_deltawave,
				     opacities.nbin,
				     T_star);
				     
}

void alfrodull_engine::correct_incident_energy(double * starflux_array_ptr,
					       bool real_star,
					       bool energy_budget_correction)
{
  printf("T_star %g, energy budget_correction: %s\n", T_star, energy_budget_correction?"true":"false" );
  if (T_star > 10 && energy_budget_correction)
    {
      dim3 grid((int(opacities.nbin) + 15 )/16, 1, 1 );
      dim3 block(16,1,1);
      
      corr_inc_energy<<<grid, block>>>(*plancktable.planck_grid,
		      starflux_array_ptr,
		      *opacities.dev_opac_deltawave,
		      real_star,
		      opacities.nbin,
		      T_star,
		      plancktable.dim);
      
      hipDeviceSynchronize();

      
    }

  // //nplanck_grid = (plancktable.dim+1)*opacities.nbin;
  // // print out planck grid for debug
  // std::unique_ptr<double[]> plgrd = std::make_unique<double[]>(plancktable.nplanck_grid);
  
  // plancktable.planck_grid.fetch(plgrd);
  // for (int i = 0; i < plancktable.nplanck_grid; i++)
  //   printf("array[%d] : %g\n", i, plgrd[i]);
}


void alfrodull_engine::set_z_calc_func( std::function<void()> & fun)
{
  calc_z_func = fun;
}

void alfrodull_engine::call_z_callback()
{
  if (calc_z_func)
    calc_z_func();
  
}
