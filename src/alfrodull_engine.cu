#include "alfrodull_engine.h"

alfrodull_engine::alfrodull_engine()
{
  printf("Creating Alfrodull engine\n");
}

void alfrodull_engine::load_opacities(const string & filename)
{
  printf("Loading opacities from %s\n", filename.c_str());

  opacities.load_opacity_table(filename);
}

void alfrodull_engine::init()
{
  printf("Alfrodull Init\n");

  load_opacities("input/opac_sample.h5");
}

void alfrodull_engine::set_parameters(const int & nlayer_,
				      const bool & iso_,
				      const double & T_star_)
{
  nlayer = nlayer_;
  ninterface = nlayer + 1;
  iso = iso_;
  T_star = T_star_;
  
  // TODO: maybe should stay in opacities object
  nbin = opacities.nbin;

  // prepare_planck_table();
}

void alfrodull_engine::allocate_internal_variables()
{
  int nlayer_nbin = nlayer*opacities.nbin;
  int nlayer_plus2_nbin = (nlayer + 2)*opacities.nbin;
  int ninterface_nbin = ninterface*opacities.nbin;
  int nlayer_wg_nbin = nlayer*opacities.ny*opacities.nbin;
  int ninterface_wg_nbin = ninterface*opacities.ny*opacities.nbin;

  // scatter cross section layer and interface
  // those are shared for print out
  printf("nlayer_nbin %d\n", nlayer_nbin);
  printf("nlayer_plus2_nbin %d\n", nlayer_plus2_nbin);
  printf("ninterface_nbin %d\n", ninterface_nbin);
  printf("nlayer_wg_nbin %d\n", nlayer_wg_nbin);
  printf("ninterface_wg_nbin %d\n", ninterface_wg_nbin);
  scatter_cross_section_lay.allocate(nlayer_nbin);
  scatter_cross_section_inter.allocate(ninterface_nbin);
  planckband_lay.allocate(nlayer_plus2_nbin);
  planckband_int.allocate(ninterface_nbin);

  
  if (iso)
    {
      delta_tau_wg.allocate(nlayer_wg_nbin);
    }
  else
    {
      delta_tau_wg_upper.allocate(nlayer_wg_nbin);
      delta_tau_wg_lower.allocate(nlayer_wg_nbin);
    }
  
  // flux computation internal quantities
  // TODO: not needed to allocate everything, depending on iso or noniso
  if (iso)
    { 
      M_term.allocate(nlayer_wg_nbin);
      N_term.allocate(nlayer_wg_nbin);   
      P_term.allocate(nlayer_wg_nbin);
      G_plus.allocate(nlayer_wg_nbin);
      G_minus.allocate(nlayer_wg_nbin);
      w_0.allocate(nlayer_wg_nbin);
    }
  else
    { 
      M_upper.allocate(nlayer_wg_nbin);
      M_lower.allocate(nlayer_wg_nbin);
      N_upper.allocate(nlayer_wg_nbin);
      N_lower.allocate(nlayer_wg_nbin);
      P_upper.allocate(nlayer_wg_nbin);
      P_lower.allocate(nlayer_wg_nbin);
      G_plus_upper.allocate(nlayer_wg_nbin);
      G_plus_lower.allocate(nlayer_wg_nbin);
      G_minus_upper.allocate(nlayer_wg_nbin);
      G_minus_lower.allocate(nlayer_wg_nbin);
      w_0_upper.allocate(nlayer_wg_nbin);
      w_0_lower.allocate(nlayer_wg_nbin);
    }

  //  dev_T_int.allocate(ninterface);

  // column mass
  // TODO: computed by grid in helios, should be computed by alfrodull or comes from THOR?
  delta_col_mass.allocate(nlayer);
  delta_col_upper.allocate(nlayer);
  delta_col_lower.allocate(nlayer);
  

  
  // dev_meanmolmass_int.allocate(ninterface);
  
  // dev_opac_wg_lay.allocate(nlayer_wg_nbin);

  // dev_trans_wg.allocate(nlayer_wg_nbin);

  // if (!iso)
  //   {
  //       dev_opac_wg_int.allocate(ninterface_wg_nbin);
  // 	dev_trans_wg_upper.allocate(nlayer_wg_nbin);
  // 	dev_trans_wg_lower.allocate(nlayer_wg_nbin);
  //   }
  
}

// return device pointers for helios data save
// TODO: how ugly can it get, really?
std::tuple<long, long, long,
	   long, long, long,
	   long, long, long,
	   long, long, long,
	   long, int, int>
alfrodull_engine::get_device_pointers_for_helios_write( )
{
  printf("Mem pointers:\n%p\n%p\n%p\n%p\n%p\n%p\n%p\n%p\n%p\n%p\n%p\n%p\n%p\n%d\n%d\n",
	 *scatter_cross_section_lay,
	 *scatter_cross_section_inter,
	 *opacities.dev_opac_interwave,
	 *opacities.dev_opac_deltawave,
	 *planckband_lay,
	 *planckband_int,
	 *plancktable.planck_grid,
	 *delta_tau_wg,
	 *delta_tau_wg_upper,
	 *delta_tau_wg_lower,
	 *delta_col_mass,
	 *delta_col_upper,
	 *delta_col_lower,
	 plancktable.dim,
	 plancktable.step);
	 
  return std::make_tuple((long) *scatter_cross_section_lay,
			 (long) *scatter_cross_section_inter,
			 (long) *opacities.dev_opac_interwave,
			 (long) *opacities.dev_opac_deltawave,
			 (long) *planckband_lay,
			 (long) *planckband_int,
			 (long) *plancktable.planck_grid,
			 (long) *delta_tau_wg,
			 (long) *delta_tau_wg_upper,
			 (long) *delta_tau_wg_lower,
			 (long) *delta_col_mass,
			 (long) *delta_col_upper,
			 (long) *delta_col_lower,
			 plancktable.dim,
			 plancktable.step);
}

// TODO: check how to enforce this: must be called after loading opacities and setting parameters
void alfrodull_engine::prepare_planck_table()
{
  plancktable.construct_planck_table(*opacities.dev_opac_interwave,
				     *opacities.dev_opac_deltawave,
				     opacities.nbin,
				     T_star);
				     
}

void alfrodull_engine::correct_incident_energy(double * starflux_array_ptr,
					       bool real_star,
					       bool energy_budget_correction)
{
  printf("T_star %g, energy budget_correction: %s\n", T_star, energy_budget_correction?"true":"false" );
  if (T_star > 10 && energy_budget_correction)
    {
      dim3 grid((int(opacities.nbin) + 15 )/16, 1, 1 );
      dim3 block(16,1,1);
      
      corr_inc_energy<<<grid, block>>>(*plancktable.planck_grid,
		      starflux_array_ptr,
		      *opacities.dev_opac_deltawave,
		      real_star,
		      opacities.nbin,
		      T_star,
		      plancktable.dim);
      
      hipDeviceSynchronize();

      
    }

  // //nplanck_grid = (plancktable.dim+1)*opacities.nbin;
  // // print out planck grid for debug
  // std::unique_ptr<double[]> plgrd = std::make_unique<double[]>(plancktable.nplanck_grid);
  
  // plancktable.planck_grid.fetch(plgrd);
  // for (int i = 0; i < plancktable.nplanck_grid; i++)
  //   printf("array[%d] : %g\n", i, plgrd[i]);
}


void alfrodull_engine::set_z_calc_func( std::function<void()> & fun)
{
  calc_z_func = fun;
}

void alfrodull_engine::call_z_callback()
{
  if (calc_z_func)
    calc_z_func();
  
}
