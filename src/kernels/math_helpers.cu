#include "hip/hip_runtime.h"
#include "math_helpers.h"


// calculates analytically the integral of the planck function
__device__ double analyt_planck(int n, double y1, double y2) {

    double dn = n;

    return exp(-dn * y2)
               * ((y2 * y2 * y2) / dn + 3.0 * (y2 * y2) / (dn * dn) + 6.0 * y2 / (dn * dn * dn)
                  + 6.0 / (dn * dn * dn * dn))
           - exp(-dn * y1)
                 * ((y1 * y1 * y1) / dn + 3.0 * (y1 * y1) / (dn * dn) + 6.0 * y1 / (dn * dn * dn)
                    + 6.0 / (dn * dn * dn * dn));
}


// calculates the power operation with a foor loop -- is allegedly faster than the implemented pow() function
__device__ double power_int(double x, int i) {

    double result = 1.0;
    int    j      = 1;

    while (j <= i) {
        result *= x;
        j++;
    }
    return result;
}
