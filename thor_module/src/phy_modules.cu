#include "hip/hip_runtime.h"
// **********************************************************************************
//
// Example of external module to reuse phy module code at various places
// This pushes the module code in another file, with a standard structure, that make it easy to
// put modules in a list and reuse them

#include "phy_modules.h"

#include "log_writer.h"

#include <math.h>
#include <memory>
#include <vector>

#include "two_streams_radiative_transfer.h"


two_streams_radiative_transfer tsrt;

// define all the modules we want to use
bool two_streams_radiative_transfer_enabled;
const bool two_streams_radiative_transfer_enabled_default = false;

// called at initialisation to store name of module used in output file, for reference.
std::string phy_modules_get_name() {
    return std::string("Alfrodull");
}

// called at initialisation for CLI reporting of configuration.
void phy_modules_print_config() {
    log::printf("  Alfrodull physics module \n");
    log::printf("   Two Stream Radiative Transfer module.\n");
    log::printf("   Module enabled: %s\n", two_streams_radiative_transfer_enabled?"True":"False");

    if (two_streams_radiative_transfer_enabled)
      tsrt.print_config();
}


// called with config_reader to add configurations keys that should be read from input file.
// use member variables to pass to config_reader as references
// config_reader will fill reference data.
// called at start, before ESP initialisation
bool phy_modules_generate_config(config_file& config_reader) {
    bool out = true;

    config_reader.append_config_var(
				    "two_streams_radiative_transfer",
				    two_streams_radiative_transfer_enabled,
				    two_streams_radiative_transfer_enabled_default);
    
    tsrt.configure(config_reader);

    return out;
}

// called at initialisation for module to allocate it's memory and cuda
// memory, receivees a device_RK_array_manager to register the arrays that
// should be updated by the RK kernels during the dynamical core Runge-Kutta step.
// Called from ESP constructor
bool phy_modules_init_mem(const ESP& esp, device_RK_array_manager& phy_modules_core_arrays) {
    // initialise all the modules memory

    bool out = true;

    if (two_streams_radiative_transfer_enabled)
      tsrt.initialise_memory(esp, phy_modules_core_arrays);

    return out;
}

// called at initialisation to initialise the state variables. Receives
// main state variables and a pointer to astorage helper object. If storage
// object is null, starting from rest, initialise the module with default.
// If storage pointer is non null, wraps the start up file used to run thor,
// it can read it's own state in that file if it has been save to and restart from there.
// Called from ESP initialisation
// after init_mem
bool phy_modules_init_data(const ESP& esp, const SimulationSetup& sim, storage* s) {
    bool out = true;
    // initialise all the modules data

    // if (s != nullptr) {
    //     // load initialisation data from storage s
    // }
    
    if (two_streams_radiative_transfer_enabled)
      out &= tsrt.initial_conditions(esp, sim, s);

    return out;
}


// called before the dynamical core step. Usually used to initialise data
// for the step or swap data from step initial state and step final state arrays.
bool phy_modules_dyn_core_loop_init(const ESP& esp) {

    return true;
}

// The arrays registered in phy_modules_init_mem are advanced in UpdateRK
// and UpdateRK2 through aRunge-Kutta scheme.
// called during slow step of dynamical core integration.
bool phy_modules_dyn_core_loop_slow_modes(const ESP&             esp,
                                          const SimulationSetup& sim,
                                          int                    nstep, // Step number
                                          double                 times) {               // Time-step [s]

    return true;
}

// called during fast step of dynamics core integrations.
bool phy_modules_dyn_core_loop_fast_modes(const ESP&             esp,
                                          const SimulationSetup& sim,
                                          int                    nstep, // Step number
                                          double                 time_step) {           // Time-step [s]

    return true;
}

// end of dynamical core loop. Used to swap initial/final state arrays before physical modules step.
bool phy_modules_dyn_core_loop_end(const ESP& esp) {

    return true;
}

//  physics integration scheme
bool phy_modules_phy_loop(ESP& esp, const SimulationSetup& sim, int nstep, double time_step) {
    // run all the modules main loop
    bool out = true;

    if (two_streams_radiative_transfer_enabled)
      tsrt.phy_loop(esp, sim, nstep, time_step);
    
    return out;
}


// called at initalisation to save parameters of physics module
bool phy_modules_store_init(storage& s) {
    // radiative transfer option
    // s.append_value(radiative_transfer_enabled ? 1.0 : 0.0,
    //                "/radiative_transfer",
    //                "-",
    //                "Using radiative transfer");

    // rt.store_init(s);

    // s.append_value(
    //     boundary_layer_enabled ? 1.0 : 0.0, "/boundary_layer", "-", "Using boundary layer");

    // bl.store_init(s);

    return true;
}

// called at end of N step to store data from integration.
bool phy_modules_store(const ESP& esp, storage& s) {

  if (two_streams_radiative_transfer_enabled)
    tsrt.store(esp, s);

    return true;
}

// called at the end of the application to free memory.
bool phy_modules_free_mem() {
    // generate all the modules config
    bool out = true;
    
    if (two_streams_radiative_transfer_enabled)
      tsrt.free_memory();

    return out;
}
